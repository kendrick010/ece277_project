#include "hip/hip_runtime.h"
#include <stdio.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h> 

#define BLOCK_SIZE 256 
#define MAX_KERNELS 16

typedef struct {
	int myPlainText;
	int myD;
	int myIsDValid; 

}IsPlainTxt;

__device__ bool isCongruent(int aNumA, int aNumB, int aModN) { 
	return aNumA % aModN == aNumB % aModN;
}
__global__ void testKeys(IsPlainTxt* aPlainTxtArr, int aXWindowSize, int aYWindowSize, int aXOffset, int aYOffset, int aN, int aCipherText) {

	// column value 
	int theXIdx = blockIdx.x;
	
	// row value
	int theYIdx = threadIdx.x + blockIdx.y * blockDim.x;

	int theD = theXIdx + aXOffset;
	int thePlainText = theYIdx +aYOffset;  
	int i = (aXWindowSize * theYIdx) + theXIdx;

	bool thePlainTextAndDCheck = (thePlainText < aCipherText) && (theD < aN); 
	bool theXYIdxCheck = i < (aXWindowSize * aYWindowSize);

	if (thePlainTextAndDCheck && theXYIdxCheck) {

		if (isCongruent(thePlainText, pow(aCipherText, theD), aN)) aPlainTxtArr[i] = { thePlainText, theD, 1 };
		else aPlainTxtArr[i] = { thePlainText, theD, 0 };

	}
	

}

void errorCatcher( hipError_t aError) { 
	if (aError != hipSuccess) {
		printf("hipMalloc failed! Error: %s\n", hipGetErrorString(aError));
		exit(EXIT_FAILURE);
	}
}

IsPlainTxt* getKeys( int aXWindowSize /*n*/, int aYWindowSize/*ciphertext*/, int aCipherText, int aN, int aXOffset, int aYOffset) {

		IsPlainTxt* theIsPlainTxtArr;
		IsPlainTxt* CUDA_theIsPlainTxtArr;
		
		int theArrSize = aXWindowSize * aYWindowSize; 
		int theArrByteSize = theArrSize * sizeof(IsPlainTxt);

		theIsPlainTxtArr=(IsPlainTxt*)malloc(theArrByteSize); 
		errorCatcher(hipMalloc((void**)&CUDA_theIsPlainTxtArr, theArrByteSize));
		
		int theNumThreads = BLOCK_SIZE; 
		
		int theNumBlockX = aXWindowSize;
		int theNumBlockY = (aYWindowSize - 1 + theNumThreads) / theNumThreads;
		
		/*
		testKeys << <dim3(theNumBlockX, theNumBlockY), theNumThreads >> > (CUDA_theIsPlainTxtArr, aCipherText, aN, 0);
		hipMemcpy(theIsPlainTxtArr, CUDA_theIsPlainTxtArr, theArrByteSize, hipMemcpyDeviceToHost);
		*/
		
		int theNumBlockYReduced = (theNumBlockY - 1 + MAX_KERNELS) / MAX_KERNELS;
		int theSegmentSize = theNumBlockX*theNumBlockYReduced*theNumThreads; 
		int thePlainTextOffset = theSegmentSize/theNumBlockX;
		
		


		hipStream_t theStreams[MAX_KERNELS];

		for (int i = 0; i < MAX_KERNELS; ++i) hipStreamCreate(&theStreams[i]);
		

		for (int i = 0; i < MAX_KERNELS; ++i) {

			int theTotalOffset = i * theSegmentSize;  

			int theYOffset = (thePlainTextOffset * i) + aYOffset;

			testKeys <<< dim3(theNumBlockX, theNumBlockYReduced), theNumThreads, 0, theStreams[i] >>>
				(CUDA_theIsPlainTxtArr+theTotalOffset, aXWindowSize, aYWindowSize, aXOffset, theYOffset , aN, aCipherText);
		} 

		
		int theSegmentByteSize = theSegmentSize * sizeof(IsPlainTxt); 
		int theRemainingBytes =  theArrByteSize - ((MAX_KERNELS - 1) * theSegmentByteSize); 
		if ((MAX_KERNELS - 1) * theSegmentSize > theArrSize) theRemainingBytes =( theArrSize - (theArrSize / theSegmentSize) * theSegmentSize) * sizeof(IsPlainTxt);


		for (int i = 0; i < MAX_KERNELS; ++i) {

			int theOffset = i * theSegmentSize; 

			if (theOffset > theArrSize) { 
				theOffset -= theSegmentSize;
				hipMemcpyAsync(theIsPlainTxtArr + theOffset, CUDA_theIsPlainTxtArr + theOffset, theRemainingBytes, hipMemcpyDeviceToHost, theStreams[i]);
				break;
			}

			int theTransferByteSize = (i != MAX_KERNELS - 1) ? theSegmentByteSize : theRemainingBytes;
			hipMemcpyAsync(theIsPlainTxtArr + theOffset, CUDA_theIsPlainTxtArr + theOffset, theTransferByteSize, hipMemcpyDeviceToHost, theStreams[i]);
		
		}

		for (int i = 0; i < MAX_KERNELS; ++i) hipStreamSynchronize(theStreams[i]);

		for (int i = 0; i < MAX_KERNELS; ++i) hipStreamDestroy(theStreams[i]);
		
		hipFree(CUDA_theIsPlainTxtArr);
		//errorCatcher(hipFree(CUDA_theIsPlainTxtArr));
		
		return theIsPlainTxtArr;
		


}

int main()
{  
	int theCipherText= 1000000; 
	int aN = 7;
	int aXWindowSize = 3; 
	int aYWindowSize = 500000;
	int aStopConditionY =(theCipherText%aYWindowSize)	? theCipherText	+ (aYWindowSize - (theCipherText % aYWindowSize)) : theCipherText;
	int aStopConditionX =(aN%aXWindowSize!=0)			? aN			+ (aXWindowSize - (aN			 % aXWindowSize)) : aN;
	for (int x = 0; x < aStopConditionX; x += aXWindowSize) {
		for (int y = 0; y < aStopConditionY; y += aYWindowSize) {
			
			IsPlainTxt* theArr = getKeys(aXWindowSize, aYWindowSize, theCipherText, aN, x, y);

			for (int j = 0; j < aYWindowSize; j++) {
				for (int k = 0; k < aXWindowSize; k++) {

					IsPlainTxt theTempStruct = theArr[(j * aXWindowSize) + k];

					printf("{%d, %d, %c}, ",
						theTempStruct.myPlainText,
						theTempStruct.myD,
						theTempStruct.myIsDValid ? 'T' : 'F');
				}
				printf("\n");
			}

			free(theArr);

		}
	}
	
	return 0;
} 

/*
typedef struct {
	int my_d_pos;
	int my_phi_n;
	int my_d;
}GCD_Tuple;


__device__  int gcd(int a_a, int a_b) {
	while (a_b != 0) {
		int the_temp = a_b;
		a_b = a_a % a_b;
		a_a = the_temp;
	}
	return a_a;
}

__global__ void calculate_gcds_via_tuples(GCD_Tuple* a_tuple_arr, int* the_d_arr, int the_d_arr_size) {

	
		//======[METHODOLOGY]======

		//1. Transfer the tuple to shared memory first

		//2. Calculate the given GCD based on the recieved tuple
	
		//3. if GCD(a,b)==1, then place a into the d array as a valid value of d

		//4. Otherwise place -1 as an invalid tuple

		//======[EXAMPLE]======

		//1. GCD_Tuple the_example {3 + 1, 3, 1} --> {relative index + d, phi(n), d};

		//2. the_d_arr[3+1] = 1 since gcd(1,3)==1;

	

__shared__ GCD_Tuple the_tuple_buffer[BLOCK_SIZE];

int i = blockDim.x * blockIdx.x + threadIdx.x;
//int the_thread_idx = threadIdx.x;

if (i < the_d_arr_size) {
	
	//the_tuple_buffer[the_thread_idx] = a_tuple_arr[i];
	//__syncthreads();


	//int the_gcd= gcd(the_tuple_buffer[the_thread_idx].my_d, the_tuple_buffer[the_thread_idx].my_phi_n);
	//if (the_gcd == 1) the_d_arr[the_tuple_buffer[the_thread_idx].my_d_pos] = the_tuple_buffer[the_thread_idx].my_d;
	//else the_d_arr[the_tuple_buffer[the_thread_idx].my_d_pos] = -1;
	
	int the_gcd = gcd(a_tuple_arr[i].my_d, a_tuple_arr[i].my_phi_n);
	if (the_gcd == 1) the_d_arr[a_tuple_arr[i].my_d_pos] = a_tuple_arr[i].my_d;
	else the_d_arr[a_tuple_arr[i].my_d_pos] = -1;


}

}


__global__ void calculate_d_tuples_2(GCD_Tuple* a_tuple_arr, int a_n_value) {
	
		//======[METHODOLOGY]======

		//1. blockIdx.x + 1 == phi(n) --> phi(n) <= n

		//2. threadIdx.x == d  --> d < phi(n)

		//	(threadIdx.x + blockDim.x*blockIdx.y is for when # of combinations exceed BLOCK_SIZE)

		//3. Now calculate relative_idx=[n(n+1) / 2] - 1 to get the beginning index to start filling in the combinations for the specific phi(n)

		//	[n(n+1)/2] is basically like n factorial, except with addition, i.e, 4+3+2+1 = 4(4+1)/2 = 10

		//4. Now insert the tuple at [relative_idx + the_d] with the [relative_idx + the_d], d and phi_n for further parallel calculation

		//======[EXAMPLE]======

		//1. n = 6

		//2. phi(n) = 1,2,3,4,5,6

		//3. d = 0,1,2,3,4,5

		//4. a_tuple_arr	=	[	0 gcd(0,1),
		//						1 gcd(0,2),  2  gcd(1,2)
		//						3 gcd(0,3),  4  gcd(1,3), 5 gcd(2,3)
		//						6 gcd(0,4),  7  gcd(1,4), 8 gcd(2,4),   9  gcd(3,4)
		//						10 gcd(0,5), 11 gcd(1,5), 12  gcd(2,5), 13 gcd(3,5), 14 gcd(4,5)
		//						15 gcd(0,6), 16 gcd(1,6), 17 gcd(2,6),  18 gcd(3,6), 19 gcd(4,6), 20 gcd(5,6) ]

	
	int the_phi_n = blockIdx.x + 1;
	int the_d = threadIdx.x + blockDim.x * blockIdx.y;

	if (the_phi_n <= a_n_value && the_d < the_phi_n) {
		int the_relative_idx = ((the_phi_n * (the_phi_n + 1)) / 2) - the_phi_n;
		a_tuple_arr[the_relative_idx + the_d] = { the_relative_idx + the_d , the_phi_n , the_d };
	}
}


int* calculate_ds_tuple(int the_n) {
	int* the_d_array;
	int* the_cuda_d_array;
	GCD_Tuple* the_cuda_tuple_array;

	int the_d_array_size = (the_n * (the_n + 1)) / 2;

	int the_d_array_byte_size = the_d_array_size * sizeof(int);

	int the_cuda_tuple_array_byte_size = the_d_array_size * sizeof(int);

	hipMalloc((void**)&the_cuda_d_array, the_d_array_byte_size);
	hipMalloc((void**)&the_cuda_tuple_array, the_cuda_tuple_array_byte_size);
	the_d_array = (int*)malloc(the_d_array_byte_size);

	int the_thread_num_a = BLOCK_SIZE;
	int the_block_num_a_x = the_n;
	// if phi(n)>BLOCK_SIZE, we will need an additional row of blocks more combinations of gcd(d, phi(n)); 
	// Example --> gcd(256, 257) is beyond BLOCK_SIZE=256, so we need to create one more row of blocks.
	int the_block_num_a_y = (the_n + the_thread_num_a - 1) / the_thread_num_a;

	calculate_d_tuples_2 << <dim3(the_block_num_a_x, the_block_num_a_y), the_thread_num_a >> > (the_cuda_tuple_array, the_n);

	int the_thread_num_b = the_d_array_size > BLOCK_SIZE ? BLOCK_SIZE : the_d_array_size;// Emulate Ceiling Division To Get Enough Blocks
	int the_block_num_b = the_d_array_size > BLOCK_SIZE ? (the_d_array_size + BLOCK_SIZE - 1) / BLOCK_SIZE : 1;
	calculate_gcds_via_tuples << <the_block_num_b, the_thread_num_b >> > (the_cuda_tuple_array, the_cuda_d_array, the_d_array_size);

	hipMemcpy(the_d_array, the_cuda_d_array, the_d_array_byte_size, hipMemcpyDeviceToHost);

	hipFree(the_cuda_d_array);
	hipFree(the_cuda_tuple_array);

	return the_d_array;
}




*/


/*
	======[OLD IMPLEMENTATION]======

	1. Uses o(2n) time complexity, because the while loops of gcd() are sequential w/ for loops.

	2. New Implementation is o(n) since there are no more for loops.

	======[OLD IMPLEMENTATION]======

		__global__ void calculate_d_tuples(GCD_Tuple* a_tuple_arr, int a_n_value) {

			int i = blockDim.x * blockIdx.x + threadIdx.x + 1;
			if (i <= a_n_value) {
				int the_relative_idx = ((i * (i + 1)) / 2) - i;
				for (int idx = 0; idx < i; idx++) {
					a_tuple_arr[the_relative_idx + idx] = { the_relative_idx + idx , i, idx };

				}
			}
		}

		int* calculate_ds_tuple_old(int the_n) {
			int* the_d_array;
			int* the_cuda_d_array;
			GCD_Tuple* the_cuda_tuple_array;

			int the_d_array_size = (the_n * (the_n + 1)) / 2;

			int the_d_array_byte_size = the_d_array_size * sizeof(int);

			int the_cuda_tuple_array_byte_size = the_d_array_size * sizeof(int);

			hipMalloc((void**)&the_cuda_d_array, the_d_array_byte_size);
			hipMalloc((void**)&the_cuda_tuple_array, the_cuda_tuple_array_byte_size);
			the_d_array = (int*)malloc(the_d_array_byte_size);

			int the_thread_num_a = the_n > BLOCK_SIZE ? BLOCK_SIZE : the_n;// Emulate Ceiling Division To Get Enough Blocks
			int the_block_num_a = the_n > BLOCK_SIZE ? (the_n + BLOCK_SIZE - 1) / BLOCK_SIZE : 1;
			calculate_d_tuples<<<the_block_num_a, the_thread_num_a>>>(the_cuda_tuple_array, the_n);

			int the_thread_num_b = the_d_array_size > BLOCK_SIZE ? BLOCK_SIZE : the_d_array_size;// Emulate Ceiling Division To Get Enough Blocks
			int the_block_num_b = the_d_array_size > BLOCK_SIZE ? (the_d_array_size + BLOCK_SIZE - 1) / BLOCK_SIZE : 1;
			calculate_gcds_via_tuples << <the_block_num_b, the_thread_num_b >> > (the_cuda_tuple_array, the_cuda_d_array, the_d_array_size);

			hipMemcpy(the_d_array, the_cuda_d_array, the_d_array_byte_size, hipMemcpyDeviceToHost);

			hipFree(the_cuda_d_array);

			return the_d_array;
		}
*/

/*
	======[OLD IMPLEMENTATION]====== 

	1. Uses o(n^2) time complexity, because the while loops of gcd() are nested in for loops. 

	2. New Implementation is o(2n) since the for loops and while loops are now sequential

	======[OLD IMPLEMENTATION]======

		__device__ void calculate_gcds(int* a_start_address, int the_amount_to_calc) {

			for (int i = 0; i < the_amount_to_calc; i++) {

				int the_gcd = gcd(i, the_amount_to_calc);
				if (the_gcd == 1) a_start_address[i] = i;
				else a_start_address[i] = -1;

			}
		}



		__global__ void calculate_d(int* a_d_array, int a_n_value)
		{
			int i = blockDim.x * blockIdx.x + threadIdx.x + 1;

			if (i <= a_n_value) {
				int* the_start_address = &a_d_array[((i * (i + 1)) / 2) - i];

				calculate_gcds(the_start_address, i);
			}

		}


		int* calculate_ds(int the_n) {

			int* the_d_array;
			int* the_cuda_d_array;

			int the_d_array_size = (the_n * (the_n + 1)) / 2;

			int the_d_array_byte_size = the_d_array_size * sizeof(int);

			hipMalloc((void**)&the_cuda_d_array, the_d_array_byte_size);
			the_d_array = (int*)malloc(the_d_array_byte_size);


			int the_thread_num = the_n > BLOCK_SIZE ? BLOCK_SIZE : the_n;

			// Emulate Ceiling Division To Get Enough Blocks
			int the_block_num = the_n > BLOCK_SIZE ? (the_n + BLOCK_SIZE - 1) / BLOCK_SIZE : 1;

			calculate_d << <the_block_num, the_thread_num >> > (the_cuda_d_array, the_n);

			hipMemcpy(the_d_array, the_cuda_d_array, the_d_array_byte_size, hipMemcpyDeviceToHost);

			hipFree(the_cuda_d_array);

			return the_d_array;

		} 
*/